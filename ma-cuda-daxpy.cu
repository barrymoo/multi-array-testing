#include "hip/hip_runtime.h"
#include <boost/multi_array.hpp>
#include <omp.h>
#include <iostream>

#define N 100000000

__global__ void daxpy(int n, double alpha, double *x, double *y) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    y[idx] += alpha * x[idx];
  }
}

int main () {
  typedef boost::multi_array<double, 1> vector;
  typedef vector::index vector_index;

  vector x(boost::extents[N]);
  vector y(boost::extents[N]);

#pragma omp parallel for
  for (vector_index i = 0; i < N; i++) {
    x[i] = 1;
    y[i] = 1;
  }

  // device malloc
  double *device_x;
  double *device_y;
  hipMalloc((void**) &device_x, N * sizeof(double));
  hipMalloc((void**) &device_y, N * sizeof(double));
  // copy to device
  hipMemcpy(&x, device_x, N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(&y, device_y, N * sizeof(double), hipMemcpyHostToDevice);
  // launch kernel
  daxpy<<<N/64,64>>>(N, 1.0, device_x, device_y);
  // copy from device
  hipMemcpy(&y, device_y, N * sizeof(double), hipMemcpyDeviceToHost);
  // Free Cuda Memory
  hipFree(device_x);
  hipFree(device_y);

  std::cout << y[0] << '\n';

  return 0;
}
