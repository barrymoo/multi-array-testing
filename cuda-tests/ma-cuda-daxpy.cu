#include "hip/hip_runtime.h"
#include <boost/multi_array.hpp>
#include <omp.h>
#include <iostream>

#define N 100000000

__global__ void daxpy_simple(int n, double alpha, double *x, double *y) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    y[idx] += alpha * x[idx];
  }
}

__global__ void daxpy(int n, double alpha, double *x, double *y) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x;
       idx < n;
       idx += blockDim.x * gridDim.x) {       
    y[idx] += alpha * x[idx];
  }
}

int main () {
  typedef boost::multi_array<double, 1> vector;
  typedef vector::index vector_index;

  vector x(boost::extents[N]);
  vector y(boost::extents[N]);

#pragma omp parallel for
  for (vector_index i = 0; i < N; i++) {
    x[i] = 1;
    y[i] = 1;
  }

  // device malloc
  double *device_x;
  double *device_y;
  hipMalloc((void**) &device_x, N * sizeof(double));
  hipMalloc((void**) &device_y, N * sizeof(double));
  // copy to device
  hipMemcpy(device_x, &x[0], N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_y, &y[0], N * sizeof(double), hipMemcpyHostToDevice);

  // Launch Kernel in Fast 
  int numSMs;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
  daxpy<<<32*numSMs,256>>>(N, 1.0, device_x, device_y);

  // copy from device
  hipMemcpy(&y[0], device_y, N * sizeof(double), hipMemcpyDeviceToHost);
  // Free Cuda Memory
  hipFree(device_x);
  hipFree(device_y);

  std::cout << y[N-1] << '\n';

  return 0;
}
